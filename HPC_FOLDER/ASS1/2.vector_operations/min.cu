#include<hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>
#define SIZE 1000
__global__ void min(int *a,int *c)
{
int i = threadIdx.x;
*c = a[0];
        if(a[i] < *c)
             {
               *c = a[i];
             }
}
int main()
{
int i;
srand(time(NULL));

int a[SIZE];
int c;
int *dev_a, *dev_c;
hipMalloc((void **) &dev_a, SIZE*sizeof(int));
hipMalloc((void **) &dev_c, SIZE*sizeof(int));
for(i=0;i<SIZE;i++)
{
a[i] = i+8;
}
hipMemcpy(dev_a,a, SIZE*sizeof(int),hipMemcpyHostToDevice);
min<<<1,SIZE>>>(dev_a,dev_c);
hipMemcpy(&c,dev_c,SIZE*sizeof(int),hipMemcpyDeviceToHost);

printf("\nmin = %d ",c);
hipFree(dev_a);
hipFree(dev_c);;
return 0;
}

