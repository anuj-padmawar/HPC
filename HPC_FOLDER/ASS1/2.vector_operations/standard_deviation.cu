#include<stdio.h>
#include<hip/hip_runtime.h>

#include<time.h>
__global__ void vecAdd(double *a,double *b,double *c,int n)
{
int id=blockIdx.x*blockDim.x+threadIdx.x;
if(id<n)
c[id]=a[id]+b[id];
}
int main()
{
srand(time(NULL));
double *h_a,*h_b,*h_c;
double *d_a,*d_b,*d_c;
int n=50;
int i=0;
clock_t t;

size_t bytes=n*sizeof(double);

h_a=(double*)malloc(bytes);
h_b=(double*)malloc(bytes);
h_c=(double*)malloc(bytes);

hipMalloc(&d_a,bytes);
hipMalloc(&d_b,bytes);
hipMalloc(&d_c,bytes);
t=clock();
for(i=0;i<n;i++)
{
h_a[i]=i;
h_b[i]=i;
}

hipMemcpy(d_a,h_a,bytes,hipMemcpyHostToDevice);
hipMemcpy(d_b,h_b,bytes,hipMemcpyHostToDevice);

int blksize,gridsize;
blksize=50;
gridsize=(int)ceil((float)n/blksize);

vecAdd<<<gridsize,blksize>>>(d_a,d_b,d_c,n);

hipMemcpy(h_c,d_c,bytes,hipMemcpyDeviceToHost);
t=clock()-t;
double time_taken=((double)t)/CLOCKS_PER_SEC;
double sum=0;
for(i=0;i<n;i++)
sum+=h_c[i];

double mean=sum/n;
double temp=0;
for(i=0;i<n;i++)
{
temp+=(h_c[i]-mean)*(h_c[i]-mean);
}
double sd=temp/n;
printf("\nAddition=%f",sum);
printf("\nAverage=%f",mean);
printf("\nSD=%f \n",sd);
 printf("\n time taken to calculate max=%f",time_taken);
hipFree(d_a);
hipFree(d_b);
hipFree(d_c);

free(h_a);
free(h_b);
free(h_c);

return(0);
}
