#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include<math.h>
#define TPB 512
#define SIZE 2048

    __global__ void max(int *a , int *c)	// kernel function definition
    {
    	int i = blockIdx.x * blockDim.x + threadIdx.x;	// initialize i to thread ID
	a[2*i] > a[2*i+1]? c[i] = a[2*i]:c[i] = a[2*i+1];
    }

    int main()
    {
    int i;
    srand(time(NULL));  //makes use of the computer's internal clock to control the choice of the seed

    int a[SIZE];
   // int c[SIZE];

    int *dev_a, *dev_c;   //GPU / device parameters

    hipMalloc((void **) &dev_a, SIZE*sizeof(int));      //assign memory to parameters on GPU
    hipMalloc((void **) &dev_c, SIZE*sizeof(int));

    for( i = 0 ; i < SIZE ; i++)
    {
    	a[i] = SIZE - i; // rand()% 1000 + 1;      // input the numbers
    }
   
        //copy the array from CPU to GPU
	
	for(i=1;i<log2((double)SIZE); i++)    
	{
		hipMemcpy(dev_a , a, SIZE*sizeof(int),hipMemcpyHostToDevice); 	
		max<<<1,SIZE/pow(2,i)>>>(dev_a,dev_c);
									// call kernel function <<<number of blocks, number of threads
    		hipMemcpy(&a, dev_c, SIZE*sizeof(int),hipMemcpyDeviceToHost);		// copy the result back from GPU to CPU
	}

	a[0] > a[1]? printf("\nMax: %d", a[0]): printf("\nMax: %d", a[1]);    
//	printf("\nmax =  %d ",a[0]);

    //cudaFree(dev_a);		// Free the allocated memory
    //cudaFree(dev_c);
  // printf("");

    return 0;
    }
