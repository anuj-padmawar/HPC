/*
 * This is a program to implement Odd-Even Sort using CUDA * It uses N/2 threads  */
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>
#include<stdlib.h>
#define N 1024

__global__ void oddeven(int *a, int flag)
{
 int index = blockIdx.x * blockDim.x + threadIdx.x;
  int temp;
  if((index >= N/2 - 1) && flag % 2 != 0) return;

  if(flag % 2 == 0) //if even phase
  {
    if(a[index *2 ] > a[index * 2 + 1])
    {
      temp = a[index * 2];
      a[index * 2] = a[index *2 +1];
      a[index * 2 +1] = temp;
    }
  }
  else { //if odd phase
    if(a[index * 2 +1 ] > a[index *2 + 2])
  {
      temp = a[index * 2 + 1];
      a[index * 2 + 1] = a[index*2+2];
      a[index*2+2] = temp;
    }
  }
}
int main()
{
 int *a;
  int *deva;
  int i;
  int size = sizeof(int) * N;
  srand((unsigned)time(NULL));

  //allocate memory in host
 a = (int *)malloc(size);

  //allocate memory in CUDA (device) memory
  hipMalloc((void **)&deva, size);

  //puting some random values in memory for generating data for sorting
  for(i=0;i<N;i++)
  {
    a[i] = rand()%N;
  }

 printf("\nNumbers before sorting: ");
  for(i=0;i<N;i++)
   {
     printf("%d ", a[i]);
   }
//recording starting time
 double start_time = clock();  

 //copy host memory data in CUDA (device) memory 
  hipMemcpy(deva, a, size, hipMemcpyHostToDevice);

  // launch a kernel N-1 times for Odd-even sort
  for(i=0;i<N;i++)
  {
    oddeven<<<N/1024, 512>>>(deva, i); //512 threads per block and total N/2/512 blocks
   }

  //copy the result back into host memory
  hipMemcpy(a, deva, size, hipMemcpyDeviceToHost);

  //Lets see the execution time
  printf("\nExecution time : %lf seconds", (clock()-start_time)/CLOCKS_PER_SEC);

  //print the result
  printf("\nOutput: ");
  for(i=0;i<N;i++)
  {
    printf("%d ", a[i]);
  }
  return 0;
}

